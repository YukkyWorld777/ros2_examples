#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "julia_set/cuda/julia_set.hpp"

#include <cmath>
#include <math.h>  // NOLINT - include .h without directory
#include <string>
#include <vector>
#include <stdio.h>

#include "hip/hip_runtime.h"  // NOLINT - include .h without directory
#include "hip/hip_runtime.h"  // NOLINT - include .h without directory


namespace 
{
__device__ float map_range( float input, float in_min, float in_max, float out_min, float out_max)
{
  return (((input - in_min) / (in_max - in_min)) * (out_max - out_min)) + out_min;
}

__device__ float3 hsv_to_rgb(float H, float S, float V) {
    float s = S/100;
    float v = V/100;
    float C = s*v;
    float X = C*(1-abs(fmod(H/60.0, 2)-1));
    float m = v-C;
    float r,g,b;
    if(H >= 0 && H < 60){
        r = C,g = X,b = 0;
    }
    else if(H >= 60 && H < 120){
        r = X,g = C,b = 0;
    }
    else if(H >= 120 && H < 180){
        r = 0,g = C,b = X;
    }
    else if(H >= 180 && H < 240){
        r = 0,g = X,b = C;
    }
    else if(H >= 240 && H < 300){
        r = X,g = 0,b = C;
    }
    else{
        r = C,g = 0,b = X;
    }

    float R = (r+m)*255;
    float G = (g+m)*255;
    float B = (b+m)*255;

	return (float3) {R, G, B};
}

__global__ void juliaset_kernel_composite(
    uint8_t * output, const uint8_t * input, const type_adaptation::julia_set::ImageMsgProperties img_properties, const type_adaptation::julia_set::JuliasetParams params)
{
    size_t x_idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    size_t x_stride = gridDim.x * blockDim.x;

    size_t y_idx = (blockDim.y * blockIdx.y) + threadIdx.y;
    size_t y_stride = gridDim.y * blockDim.y;

    for(size_t row = y_idx; row < img_properties.height; row += y_stride) {
        for(size_t col = x_idx; col < img_properties.width; col += x_stride) {
            size_t color_idx = (row * img_properties.row_step) + (col * img_properties.color_step);

            // Map height and width on a scale of -2 to 2
            float real_part = map_range(col, params.kMinColRange, params.kMaxColRange, params.kMinXRange, params.kMaxXRange);
            float img_part = map_range(row, params.kMinRowRange, params.kMaxRowRange, params.kMinYRange, params.kMaxYRange);
            float orig_real_part = params.kStartX * cos(params.kCurrentAngle);
            float orig_img_part = params.kStartY * sin(params.kCurrentAngle);
            float new_real_part, new_img_part;
            size_t counter = 0;
            while(counter < params.kMaxIterations)
            {
                new_real_part = (real_part * real_part) - (img_part * img_part);
                new_img_part = 2 * real_part * img_part;
                
                if((real_part * real_part + img_part * img_part) > params.kBoundaryRadius * params.kBoundaryRadius)
                    break;
                real_part = new_real_part + orig_real_part;
                img_part = new_img_part + orig_img_part;
                counter++;
            }

            if(counter == params.kMaxIterations) {
                output[color_idx + img_properties.red_offset] = input[color_idx + img_properties.red_offset] / 4;
                output[color_idx + img_properties.green_offset] = input[color_idx + img_properties.green_offset] / 4;
                output[color_idx + img_properties.blue_offset] = input[color_idx + img_properties.blue_offset] / 4;
            } else {

                float h = fmod((counter * 360 / params.kMaxIterations), 360);
                float s = 100;
                float v = pow((float)counter / params.kMaxIterations,.020)*100;
                float3 color = hsv_to_rgb(h,s,v);
                output[color_idx + img_properties.red_offset] =  color.x - input[color_idx + img_properties.red_offset] / 16;
                output[color_idx + img_properties.green_offset] =  color.y - input[color_idx + img_properties.green_offset] / 16;
                output[color_idx + img_properties.blue_offset] =   color.z - input[color_idx + img_properties.blue_offset] / 16;
            }
        }
    }
}

__global__ void map_kernel(
    float * output, const type_adaptation::julia_set::ImageMsgProperties img_properties, const type_adaptation::julia_set::JuliasetParams params)
{
    size_t x_idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    size_t x_stride = gridDim.x * blockDim.x;

    size_t y_idx = (blockDim.y * blockIdx.y) + threadIdx.y;
    size_t y_stride = gridDim.y * blockDim.y;

    const uint8_t kChannel = 3;

    for(size_t row = y_idx; row < img_properties.height; row += y_stride) {
        for(size_t col = x_idx; col < img_properties.width; col += x_stride) {
            size_t x_idx = (row * img_properties.width * kChannel) + (col * kChannel);
            size_t y_idx = x_idx + 1;
            size_t z_idx = y_idx + 1;

            output[x_idx] = map_range(col, params.kMinColRange, params.kMaxColRange, params.kMinXRange, params.kMaxXRange);
            output[y_idx] = map_range(row, params.kMinRowRange, params.kMaxRowRange, params.kMinYRange, params.kMaxYRange);
            output[z_idx] = 0.0;
        }
    }
}

__global__ void juliaset_kernel(size_t curr_iteration,
    float * output, const float * input, const type_adaptation::julia_set::ImageMsgProperties img_properties, const type_adaptation::julia_set::JuliasetParams params)
{
    size_t x_idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    size_t x_stride = gridDim.x * blockDim.x;

    size_t y_idx = (blockDim.y * blockIdx.y) + threadIdx.y;
    size_t y_stride = gridDim.y * blockDim.y;

    const uint8_t kChannel = 3;

    for(size_t row = y_idx; row < img_properties.height; row += y_stride) {
        for(size_t col = x_idx; col < img_properties.width; col += x_stride) {
            size_t x_idx = (row * img_properties.width * kChannel) + (col * kChannel);
            size_t y_idx = x_idx + 1;
            size_t z_idx = y_idx + 1;

            float real_part = input[x_idx];
            float img_part = input[y_idx];
            float orig_real_part = params.kStartX * cos(params.kCurrentAngle);
            float orig_img_part = params.kStartY * sin(params.kCurrentAngle);
            float new_real_part, new_img_part;
          
            if(input[z_idx] == 0.0){
                if((real_part * real_part + img_part * img_part) > params.kBoundaryRadius * params.kBoundaryRadius) {
                    output[z_idx] = 1.0 + curr_iteration;
                    return;
                }
                new_real_part = (real_part * real_part) - (img_part * img_part);
                new_img_part = 2 * real_part * img_part;
                
                output[x_idx] = new_real_part + orig_real_part;
                output[y_idx] = new_img_part + orig_img_part;
                output[z_idx] = 0.0;
            } else {
                return;
            }
        }
    }
}

__global__ void colorize_kernel(
    uint8_t * output, const float * input, const type_adaptation::julia_set::ImageMsgProperties img_properties, const type_adaptation::julia_set::JuliasetParams params)
{
    size_t x_idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    size_t x_stride = gridDim.x * blockDim.x;

    size_t y_idx = (blockDim.y * blockIdx.y) + threadIdx.y;
    size_t y_stride = gridDim.y * blockDim.y;

    const uint8_t kChannel = 3;

    for(size_t row = y_idx; row < img_properties.height; row += y_stride) {
        for(size_t col = x_idx; col < img_properties.width; col += x_stride) {
            
            size_t x_idx = (row * img_properties.width * kChannel) + (col * kChannel);
            size_t y_idx = x_idx + 1;
            size_t z_idx = y_idx + 1;
            
            size_t color_idx = (row * img_properties.row_step / sizeof(4)) + (col * img_properties.color_step);

            if(input[z_idx] == (float)0.0) {
                output[color_idx + img_properties.red_offset] = input[x_idx] / 4;
                output[color_idx + img_properties.green_offset] = input[y_idx] / 4;
                output[color_idx + img_properties.blue_offset] = input[z_idx] / 4;
            } else {

                float h = fmod(((input[z_idx] - 1) * 360 / params.kMaxIterations), 360);
                float s = 100;
                float v = pow((float)(input[z_idx] - 1) / params.kMaxIterations,.020)*100;
                float3 color = hsv_to_rgb(h,s,v);
                output[color_idx + img_properties.red_offset] =  color.x - input[x_idx] / 16;
                output[color_idx + img_properties.green_offset] =  color.y - input[y_idx] / 16;
                output[color_idx + img_properties.blue_offset] =   color.z - input[z_idx] / 16;
            }
        }
    }
}

}  // namespace

namespace type_adaptation
{
namespace julia_set
{
Juliaset::Juliaset(ImageMsgProperties img_properties, JuliasetParams parameters): 
 image_msg_property_{img_properties},
 parameters_{parameters} {configure_kernel_execution();}

void Juliaset::configure_kernel_execution() {
    // Get the number of CUDA blocks & threads
    size_t num_blocks_x = (image_msg_property_.width + num_threads_per_block_x_ - 1) / 
                        num_threads_per_block_x_;
    size_t num_blocks_y = (image_msg_property_.height + num_threads_per_block_y_ - 1) /
                        num_threads_per_block_y_;

    num_of_blocks_ = dim3(num_blocks_x, num_blocks_y, 1);
    threads_per_block_ = dim3(num_threads_per_block_x_, num_threads_per_block_y_, 1);
}

void Juliaset::compute_juliaset_composite(float & current_angle, u_int8_t * image, const hipStream_t & stream)
{
    parameters_.kCurrentAngle = current_angle;
    // Invoke CUDA kernel
    juliaset_kernel_composite<<<num_of_blocks_, threads_per_block_, 0, stream>>>(image,
                                                                            image,
                                                                            image_msg_property_,
                                                                            parameters_); 
}

void Juliaset::map(float * out_mat, const hipStream_t & stream)
{
    // Invoke CUDA kernel
    map_kernel<<<num_of_blocks_, threads_per_block_, 0, stream>>>(out_mat,
                                                                image_msg_property_,
                                                                parameters_);
}

void Juliaset::compute_juliaset_pipeline(
    size_t curr_iteration, float & current_angle, float * image, const hipStream_t & stream)
{
    parameters_.kCurrentAngle = current_angle;
    // Invoke CUDA kernel
    juliaset_kernel<<<num_of_blocks_, threads_per_block_, 0, stream>>>(curr_iteration,
                                                                    image,
                                                                    image,
                                                                    image_msg_property_,
                                                                    parameters_);
  
}

void Juliaset::colorize(
    uint8_t * output, const float * input, const hipStream_t & stream)
{
    // Invoke CUDA kernel
    colorize_kernel<<<num_of_blocks_, threads_per_block_, 0, stream>>>(output,
                                                                    input,
                                                                    image_msg_property_,
                                                                    parameters_);
}

}  // namespace julia_set
}  // namespace type_adaptation
